#include "hip/hip_runtime.h"
#include<stdio.h>
#include <iostream>
#include <assert.h>
#include <unistd.h>
#include "mpi.h"
#include "nvshmem.h"
#include "nvshmemx.h"

#include "../util/error_util.cuh"
#include "../util/nvshmem_util.cuh"

namespace dev {
	template<typename LOCAL_T, typename RECV_T, typename COUNTER_T, int PADDING_SIZE>
	class Queues {
	public:
		int n_pes;
		int my_pe;
		int group_size;
	
		COUNTER_T recv_capacity;
		COUNTER_T local_capacity;
			
		LOCAL_T *local_queues;
		RECV_T *recv_queues;
		RECV_T *agg_queues;	

		int num_agg_queues;
		int num_local_queues;
		int total_num_queues;

		volatile COUNTER_T *start, *start_alloc, *end, *end_alloc, *end_max, *end_count;
	    volatile int *stop;
		Queues(int _n_pes, int _my_pe, int _group_size, LOCAL_T *l_q, RECV_T *r_q, RECV_T *a_q, COUNTER_T l_capacity, COUNTER_T r_capacity,
    	       COUNTER_T *_start, COUNTER_T *_start_alloc, COUNTER_T *_end_alloc, COUNTER_T *_end, COUNTER_T *_end_max, COUNTER_T *_end_count, int *_stop, int _num_l_q,
    	       int _num_a_q): n_pes(_n_pes), my_pe(_my_pe), group_size(_group_size), local_queues(l_q), recv_queues(r_q), agg_queues(a_q), 
    	       num_local_queues(_num_l_q), num_agg_queues(_num_a_q), local_capacity(l_capacity), recv_capacity(r_capacity) 
    	{
    	    start = (volatile COUNTER_T *)_start;
    	    start_alloc = (volatile COUNTER_T *)_start_alloc;
    	    end_alloc = (volatile COUNTER_T *)_end_alloc;
    	    end = (volatile COUNTER_T *)_end;
    	    end_max = (volatile COUNTER_T *)_end_max;
    	    end_count = (volatile COUNTER_T *)_end_count;
    	    stop = (volatile int *)_stop;

    	    total_num_queues=num_local_queues+n_pes-1+num_agg_queues;
    	}
	};
}

template<typename LOCAL_T, typename RECV_T, typename COUNTER_T, int PADDING_SIZE>
class Queues {
public:
	int n_pes;
	int my_pe;
	int nodes_size;
	int node_id;
	int group_size;
	int group_id;

	COUNTER_T recv_capacity;
	COUNTER_T local_capacity;
		
	LOCAL_T *local_queues;
	RECV_T *recv_queues;
	RECV_T *agg_queues;	

	int num_agg_queues;
	int num_local_queues;
	int total_num_queues;

	COUNTER_T *counters;
	int num_counters=7;
	COUNTER_T *start, *start_alloc, *end, *end_alloc, *end_max, *end_count;
    int *stop;

	Queues() {}
    ~Queues() { release(); }

    __host__ void baseInit(int _n_pes, int _my_pe, int _group_id, int _group_size, int local_id, int local_size,
    COUNTER_T l_capacity, COUNTER_T r_capacity, int l_queues = 1, bool PRINT_INFO = false)
    {
        n_pes = _n_pes;
        my_pe = _my_pe;
        nodes_size= _group_size;
        node_id = _group_id;
        group_size = local_size;
        group_id = local_id;

        local_capacity = l_capacity;
        recv_capacity = r_capacity;

        num_local_queues = l_queues;
        num_agg_queues = (nodes_size-1)*group_size;
        total_num_queues = num_local_queues+n_pes-1+num_agg_queues;

        alloc(PRINT_INFO);
    }

	dev::Queues<LOCAL_T, RECV_T, COUNTER_T, PADDING_SIZE>
    deviceObject() const {
    	return dev::Queues<LOCAL_T, RECV_T, COUNTER_T, PADDING_SIZE>
        (n_pes, my_pe, group_size, local_queues,  recv_queues, agg_queues, local_capacity, recv_capacity,
        start, start_alloc, end_alloc, end, end_max, end_count, stop, num_local_queues, num_agg_queues);
    }

private:
	void alloc(bool PRINT_INFO = false)
    {
        if(local_capacity+recv_capacity <= 0) return;
        if(PRINT_INFO)
                std::cout << "pe "<< my_pe << " called distributed queue base allocator\n";
        CUDA_CHECK(hipMalloc(&local_queues, sizeof(LOCAL_T)*local_capacity*num_local_queues));
        CUDA_CHECK(hipMemset(local_queues, 0xffffffff, sizeof(LOCAL_T)*local_capacity*num_local_queues));

        recv_queues = (RECV_T *)nvshmem_malloc(sizeof(RECV_T)*recv_capacity*(n_pes-1));
        CUDA_CHECK(hipMemset(recv_queues, 0xffffffff, sizeof(RECV_T)*recv_capacity*(n_pes-1)));

        agg_queues = (RECV_T *)nvshmem_malloc(sizeof(RECV_T)*recv_capacity*num_agg_queues);
        CUDA_CHECK(hipMemset(agg_queues, 0xffffffff, sizeof(RECV_T)*recv_capacity*num_agg_queues));

        counters = (COUNTER_T *)nvshmem_malloc(sizeof(COUNTER_T)*num_counters*PADDING_SIZE*total_num_queues);
        start = counters;
        start_alloc = (counters+1*PADDING_SIZE*total_num_queues);
        end_alloc = (counters+2*PADDING_SIZE*total_num_queues);
        end = (counters+3*PADDING_SIZE*total_num_queues);
        end_max = (counters+4*PADDING_SIZE*total_num_queues);
        end_count = (counters+5*PADDING_SIZE*total_num_queues);

        stop = (int *)(counters+6*PADDING_SIZE*total_num_queues);
    }

	void release(bool PRINT_INFO = false) {
        if(local_capacity+recv_capacity <= 0) return;
        if(PRINT_INFO)
            std::cout << "pe "<< my_pe << " call distributed queue base destructor\n";
        if(local_queues!=NULL)
        CUDA_CHECK(hipFree(local_queues));

		// if uncommend following lines, got errors: src/util/cs.cpp:26: non-zero status: 22: No such file or directory, exiting... mutex lock failed
        //nvshmem_free(agg_queues);
        //nvshmem_free(recv_queues);
        //nvshmem_free(counters);
    }
};

template<typename T, typename Y>
struct Pack {
	T id;
	Y value;
	__device__ __host__ Pack() {}
	__device__ __host__ Pack(T _id, Y _value): id(_id), value(_value) {}
};

template<typename LOCAL_T, typename RECV_T, typename COUNTER_T, int PADDING_SIZE>
__global__ void set(dev::Queues<LOCAL_T, RECV_T, COUNTER_T, PADDING_SIZE> queue) {
	if(threadIdx.x == 0)
	nvshmem_uint32_p((uint32_t *)(queue.end+queue.num_local_queues*PADDING_SIZE), 12314, (queue.my_pe^1));
}

int main(int argc, char** argv)
{

    int n_pes, my_pe, group_id, group_size, local_id, local_size; 
    nvshm_mpi_init(my_pe, n_pes, group_id, group_size, local_id, local_size, &argc, &argv);

	Queues<int, Pack<int, float>, uint32_t, 32> queue;
	queue.baseInit(n_pes, my_pe, group_id, group_size, local_id, local_size, 1024, 300000000);
	
	hipStream_t stream;
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	nvshmem_barrier_all();
	
	set<<<1,32, 0, stream>>>(queue.deviceObject());

	CUDA_CHECK(hipStreamSynchronize(stream));
	nvshmem_barrier_all();

	uint32_t end;
	CUDA_CHECK(hipMemcpy(&end, queue.end+32, sizeof(uint32_t), hipMemcpyDeviceToHost));
	printf("PE %d, end %d\n", my_pe, end);

	nvshmem_barrier_all();
    nvshm_mpi_finalize();
	return 0;
}
